#include <iostream>
#include <time.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"


__global__ void add(int *a, int *b, int *c)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  while(index < 20)
  {
    c[index] =  a[index] + b[index];
    index += blockDim.x * gridDim.x;
    printf("%d\n",gridDim.x);
  }
}

int main()
{
  int n = 20;
  int *a = (int *)malloc(sizeof(int)*n);
  int *b = (int *)malloc(sizeof(int)*n);
  int *c = (int *)malloc(sizeof(int)*n);

  int *dev_a;
  int *dev_b;
  int *dev_c;
  int *dev_n;

  // 
  for(int i=0;i<n;i++)
  {
    a[i] = i;
    b[i] = i;
  }

  std::cout << ">>>> 1 <<<<" <<std::endl;

  hipMalloc((void**)&dev_a, sizeof(int)*n);
  hipMalloc((void**)&dev_b, sizeof(int)*n);
  hipMalloc((void**)&dev_c, sizeof(int)*n);
  hipMalloc((void**)&dev_n, sizeof(int));
  // add<<<1,1>>>(2,7,dev_c);

  std::cout << ">>>> 2 <<<<" <<std::endl;

  hipMemcpy(dev_a,a,sizeof(int)*n,hipMemcpyHostToDevice);
  hipMemcpy(dev_b,b,sizeof(int)*n,hipMemcpyHostToDevice);
  hipMemcpy(dev_n,&n,sizeof(int),hipMemcpyHostToDevice);

  std::cout << ">>>> 3 <<<<" <<std::endl;

  dim3 blockSize(20);
  dim3 gridSize((n+blockSize.x-1)/blockSize.x); //1
  // add<<<grid，block>>>
  // gridSize個線程塊*blockSize個線程/線程塊
  // add<<<gridSize, blockSize>>>(dev_a,dev_b,dev_c);
  add<<<1, 1>>>(dev_a,dev_b,dev_c);

  std::cout << ">>>> 4 <<<<" <<std::endl;

  hipMemcpy(c,dev_c,sizeof(int)*n,hipMemcpyDeviceToHost);

  for(int i=0;i<n;i++)
  {
    printf("c[%d] = %d\n",i,c[i]);
  }
  
  hipFree(dev_c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_n);
  
  free(a);
  free(b);
  free(c);

  std::cout << ">>>> 5 <<<<" <<std::endl;

  return 0;
}
