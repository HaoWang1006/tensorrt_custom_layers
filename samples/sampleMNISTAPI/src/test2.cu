#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <malloc.h>
#include <random>
#include <time.h>

const int threadPerBlock = 16;

using namespace std;

texture<int> texA;
texture<int> texB;

// hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

hipError_t mulWithCuda(const int *a, const int *b, int *result);

// hipError_t mulWithCudaTex(const int *a, const int *b, int *result, const int M, const int N, const int S);

__global__ void addKernel(int *c, const int *a, const int *b)
{
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}

/* MatMultiply：CPU下矩阵乘法
*  a:第一个矩阵指针，表示a[M][N];
*  b:第二个矩阵指针，表示b[N][S];
*  result:结果矩阵，表示为result[M][S];
*/
void CPUMatMultiply(const int *a, const int *b, int *result, const int M, const int N, const int S)
{
  for (int i = 0; i < M; i++)
  {
    for (int j = 0; j < S; j++)
    {
      int index = i * S + j;
      result[index] = 0;

      //计算每一个元素的结果
      for (int k = 0; k < N; k++)
      {
        result[index] += a[i * N + k] * b[k * S + j];
      }
    }
  }
}

/* gpuMatMultKernel：GPU下矩阵乘法核函数
*  a:第一个矩阵指针，表示a[M][N]
*  b:第二个矩阵指针，表示b[N][S]
*  result:结果矩阵，表示result[M][S]
*/
__global__ void gpuMatMultKernel(const int *a, const int *b, int *result, const int M, const int N, const int S)
{
  //int threadId = threadIdx.x + blockIdx.x * blockDim.x;

  int threadId = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;

  // int row = threadId.y + blockIdx.y * blockDim.y;
  // int col = threadId.x + blockIdx.x * blockDim.x;
  // int threadId = row * col;

  if (threadId < M * S)
  {
    int row = threadId / S;
    int column = threadId % S;
    // printf("gridDim.x === %d, blockDim.x === %d\n", row, column);
    result[threadId] = 0;
    for (int i = 0; i < N; i++)
    {
      result[threadId] += a[row * N + i] * b[i * S + column];
    }
  }
}

/* gpuMatMultWithSharedKernel：GPU下使用shared内存的矩阵乘法
*  a:第一个矩阵指针，表示a[height_A][width_A]
*  b:第二个矩阵指针，表示b[width_A][width_B]
*  result:结果矩阵，表示result[height_A][width_B]
*/
template <int BLOCK_SIZE>
__global__ void gpuMatMultWithSharedKernel(const int *a, const int *b, int *result, const int height_A, const int width_A, const int width_B)
{
  int block_x = blockIdx.x;
  int block_y = blockIdx.y;
  int thread_x = threadIdx.x;
  int thread_y = threadIdx.y;

  if ((thread_y + block_y * blockDim.y) * width_B + block_x * blockDim.x + thread_x >= height_A * width_B)
  {
    return;
  }

  // printf(" === %d\n",width_A);
  const int begin_a = blockIdx.y * blockDim.y * width_A;
  const int end_a = begin_a + width_A - 1;
  const int step_a = blockDim.x;
  printf("@@@@@@@1 %d,%d,%d\n",begin_a,end_a,step_a);
  printf("@@@@@@@1-1 %d\n",gridDim.x);

  const int begin_b = blockIdx.x * blockDim.x;
  const int step_b = blockDim.y * width_B;
  printf("@@@@@@@2 %d,%d\n",begin_b,step_b);

  int result_temp = 0;
  int index_a = begin_a;
  int index_b = begin_b;
  printf("@@@@@@@3 %d,%d\n", blockIdx.y,blockIdx.x);
  for (; index_a < end_a; index_a += step_a, index_b += step_b)
  {
    __shared__ int SubMat_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int SubMat_B[BLOCK_SIZE][BLOCK_SIZE];
    printf("ttttt = %d, %d,%d,%d,%d\n",index_a,index_b,step_a,step_b,end_a);
    printf("%d,%d,%d,%d\n",thread_y,thread_x,a[index_a + threadIdx.y * width_A + threadIdx.x],b[index_b + thread_y * width_B + thread_x]);
    // 矩阵A向右走，step为blockdim.x
    SubMat_A[thread_y][thread_x] = a[index_a + threadIdx.y * width_A + threadIdx.x];
    // 矩阵B向下走，step为blockdim.y*width_B
    SubMat_B[thread_y][thread_x] = b[index_b + threadIdx.y * width_B + threadIdx.x];
    printf("$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$\n");
    __syncthreads();
    
    printf("test time");
    for (int i = 0; i < BLOCK_SIZE; i++)
    {
      // 矩阵A变x,即thread_x; 矩阵B变y, 即thread_y
      printf("------ %d,%d,%d,%d,%d\n",i,thread_y,thread_x,SubMat_A[thread_y][i],SubMat_B[i][thread_x]);
      result_temp += SubMat_A[thread_y][i] * SubMat_B[i][thread_x];
      printf("%d,%d@@@@@@@@@@@@@@@%d\n",thread_y,thread_x,result_temp);
    }
    __syncthreads();
  }

  int begin_result = blockIdx.y * blockDim.y * width_B + begin_b;
  result[begin_result + thread_y * width_B + thread_x] = result_temp;
  printf("%d\n",result[begin_result + thread_y * width_B + thread_x]);
}

/* gpuMatMultWithTextureKernel：GPU下使用texture内存的矩阵乘法
*  result：结果矩阵，表示为result[M][S];
*  M：表示为矩阵A与矩阵result的行数
*  N：表示矩阵A的列数，矩阵B的行数
*  S：表示矩阵B和矩阵result的列数
*/
// __global__ void gpuMatMultWithTextureKernel(int *result, const int M, const int N, const int S)
// {
//   int x = threadIdx.x + blockIdx.x * blockDim.x;
//   int y = threadIdx.y + blockIdx.y * blockDim.y;
//   int offset = x + y * blockDim.x * gridDim.x;

//   if (offset < M * S)
//   {
//     int a = 0, b = 0;
//     int temp_result = 0;
//     for (int i = 0;                     i < N; i++)
//     {
//       a = tex1Dfetch(texA, y * N + i);
//       b = tex1Dfetch(texB, i * S + x);
//       temp_result += a * b;
//     }
//     result[offset] = temp_result;
//   }
// }

// main主函数，分别运行CPU和GPU矩阵乘法函数，比较二者的运行时间
int main()
{
  //确定矩阵的大小
  int M = 4, N = 4, S = 4;

  //分配矩阵空间
  int *a = (int *)malloc(M * N * sizeof(int));
  if (NULL == a)
  {
    printf("the malloc of Mat a is failed!\n");
    return 0;
  }

  int *b = (int *)malloc(N * S * sizeof(int));
  if (NULL == b)
  {
    printf("the malloc of Mat b is failed!\n");
    return 0;
  }

  //cpu与gpu的结果矩阵分别存放
  int *cpuResult = (int *)malloc(M * S * sizeof(int));
  if (NULL == cpuResult)
  {
    printf("the malloc of Mat cpuResult is failed!\n");
    return 0;
  }

  int *gpuResult = (int *)malloc(M * S * sizeof(int));
  if (NULL == cpuResult)
  {
    printf("the malloc of Mat gpuResult is failed!\n");
    return 0;
  }

  //生成矩阵数据
  printf("\nstart random the Mat a...\n");
  for (int i = 0; i < M; i++)
  {
    for (int j = 0; j < N; j++)
    {
      a[i * N + j] = rand() % 50;
      printf("%d  ",a[i * N + j]);
    }
    printf("\n");
  }

  printf("\nstart random the Mat b...\n");
  for (int i = 0; i < N; i++)
  {
    for (int j = 0; j < S; j++)
    {
      b[i * S + j] = rand() % 50;
      printf("%d  ",b[i * S + j]);
    }
    printf("\n");
  }

  //统计CPU运行乘法的时间
  clock_t start, finish;
  double totalTime = 0.0;
  start = clock();

  //调用CPU矩阵乘法函数
  CPUMatMultiply(a, b, cpuResult, M, N, S);

  finish = clock();
  totalTime = (double)(finish - start) / CLOCKS_PER_SEC;
  printf("\nThe total time is %lf seconds!\n", totalTime);

  //调用GPU矩阵乘法函数
  hipError_t cudaStatus = mulWithCuda(a, b, gpuResult);
  //hipError_t cudaStatus = mulWithCudaTex(a, b, gpuResult, M, N, S);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "mulWithCuda failed!");
    return 0;
  }

  //打印结果矩阵result
  /*printf("\nthe result of CPU :\n");
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			printf("%d\t", cpuResult[i * M + j]);
		}
		printf("\n");
	}
	printf("\nthe result of GPU :\n");
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			printf("%d\t", gpuResult[i * M + j]);
		}
		printf("\n");
	}*/

  //确认CPU和GPU矩阵乘法结果是否相同，从而说明结果是否正确
  for (int i = 0; i < M; i++)
  {
    for (int j = 0; j < S; j++)
    {
      if (cpuResult[i * M + j] != gpuResult[i * M + j])
      {
        printf("the Result isn't equal!\n");
        return 0;
      }
    }
  }

  return 0;
}

// // Helper function for using CUDA to add vectors in parallel.
// hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
// {
//   int *dev_a = 0;
//   int *dev_b = 0;
//   int *dev_c = 0;
//   hipError_t cudaStatus;

//   // Choose which GPU to run on, change this on a multi-GPU system.
//   cudaStatus = hipSetDevice(0);
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//     goto Error;
//   }

//   // Allocate GPU buffers for three vectors (two input, one output)    .
//   cudaStatus = hipMalloc((void **)&dev_c, size * sizeof(int));
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "hipMalloc failed!");
//     goto Error;
//   }

//   cudaStatus = hipMalloc((void **)&dev_a, size * sizeof(int));
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "hipMalloc failed!");
//     goto Error;
//   }

//   cudaStatus = hipMalloc((void **)&dev_b, size * sizeof(int));
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "hipMalloc failed!");
//     goto Error;
//   }

//   // Copy input vectors from host memory to GPU buffers.
//   cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "hipMemcpy failed!");
//     goto Error;
//   }

//   cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "hipMemcpy failed!");
//     goto Error;
//   }

//   // Launch a kernel on the GPU with one thread for each element.
//   addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

//   // Check for any errors launching the kernel
//   cudaStatus = hipGetLastError();
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//     goto Error;
//   }

//   // hipDeviceSynchronize waits for the kernel to finish, and returns
//   // any errors encountered during the launch.
//   cudaStatus = hipDeviceSynchronize();
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//     goto Error;
//   }

//   // Copy output vector from GPU buffer to host memory.
//   cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "hipMemcpy failed!");
//     goto Error;
//   }

// Error:
//   hipFree(dev_c);
//   hipFree(dev_a);
//   hipFree(dev_b);

//   return cudaStatus;
// }

// 调用CUDA运行GPU矩阵乘法核函数
hipError_t mulWithCuda(const int *a, const int *b, int *result)
{
  int *dev_a;
  int *dev_b;
  int *dev_result;

  const int M = 4;
  const int N = 4;
  const int S = 4;

  hipError_t cudaStatus;

  cudaStatus = hipSetDevice(0);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
    // goto Error;
  }

  cudaStatus = hipMalloc((void **)&dev_a, M * N * sizeof(int));
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMalloc dev_a failed!\n");
    // goto Error;
  }

  cudaStatus = hipMalloc((void **)&dev_b, N * S * sizeof(int));
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMalloc dev_b failed!\n");
    // goto Error;
  }

  cudaStatus = hipMalloc((void **)&dev_result, M * S * sizeof(int));
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMalloc dev_result failed!\n");
    // goto Error;
  }

  cudaStatus = hipMemcpy(dev_a, a, M * N * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "cudamemcpy dev_a failed!\n");
    // goto Error;
  }

  cudaStatus = hipMemcpy(dev_b, b, N * S * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMemcpy dev_b failed!\n");
    // goto Error;
  }

  hipEvent_t gpuStart, gpuFinish;
  float elapsedTime;
  hipEventCreate(&gpuStart);
  hipEventCreate(&gpuFinish);
  hipEventRecord(gpuStart, 0);

  /*const int THREADNUM = 256;
	const int BLOCKNUM = (M * S + 255) / 256;*/

  const int BLOCK_SIZE = 2;
  dim3 block(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid((S + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
  //gpuMatMultKernel<<<grid, block>>>(dev_a, dev_b, dev_result, M, N, S);
  gpuMatMultWithSharedKernel<BLOCK_SIZE><<<grid, block>>>(dev_a, dev_b, dev_result, M, N, S);

  hipEventRecord(gpuFinish, 0);
  hipEventSynchronize(gpuFinish);
  hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
  printf("\nThe runing time of GPU on Mat Multiply is %f seconds.\n", elapsedTime / 1000.0);

  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));
    // goto Error;
  }

  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipDeviceSynchronize return Error code %d after Kernel launched!\n", cudaStatus);
    // goto Error;
  }

  cudaStatus = hipMemcpy(result, dev_result, M * S * sizeof(int), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMemcpy result failed!\n");
    // goto Error;
  }

  // Error:
  //   hipFree(dev_a);
  //   hipFree(dev_b);
  //   hipFree(dev_result);

  return cudaStatus;
}

// //调用CUDA运行GPU矩阵乘法核函数
// //将矩阵A与矩阵B绑定到纹理内存中
// hipError_t mulWithCudaTex(const int *a, const int *b, int *result, const int M, const int N, const int S)
// {
//   int *dev_a = 0;
//   int *dev_b = 0;
//   int *dev_result = 0;

//   hipError_t cudaStatus;

//   cudaStatus = hipSetDevice(0);
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "hipSetDevice failed! Do you have a CUDA_capable GPU installed?\n");
//     goto Error;
//   }

//   cudaStatus = hipMalloc((void **)&dev_a, M * N * sizeof(int));
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "hipMalloc dev_a failed!\n");
//     goto Error;
//   }

//   cudaStatus = hipMalloc((void **)&dev_b, N * S * sizeof(int));
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "hipMalloc dev_b failed!\n");
//     goto Error;
//   }

//   cudaStatus = hipMalloc((void **)&dev_result, M * S * sizeof(int));
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "hipMalloc dev_result failed!\n");
//     goto Error;
//   }

//   hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
//   cudaStatus = hipBindTexture(NULL, texA, dev_a, desc, M * N * sizeof(int));
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "hipBindTexture texA failed!\n");
//     goto Error;
//   }

//   cudaStatus = hipBindTexture(NULL, texB, dev_b, desc, N * S * sizeof(int));
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "hipBindTexture texB failed!\n");
//     goto Error;
//   }

//   cudaStatus = hipMemcpy(dev_a, a, M * N * sizeof(int), hipMemcpyHostToDevice);
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "cudamemcpy dev_a failed!\n");
//     goto Error;
//   }

//   cudaStatus = hipMemcpy(dev_b, b, N * S * sizeof(int), hipMemcpyHostToDevice);
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "hipMemcpy dev_b failed!\n");
//     goto Error;
//   }

//   hipEvent_t gpuStart, gpuFinish;
//   float elapsedTime;
//   hipEventCreate(&gpuStart);
//   hipEventCreate(&gpuFinish);
//   hipEventRecord(gpuStart, 0);

//   const int BLOCK_SIZE = 16;
//   if ((M % BLOCK_SIZE != 0) && (S % BLOCK_SIZE != 0))
//   {
//     fprintf(stderr, "M or S can't be dividen by 16!\n");
//     goto Error;
//   }

//   dim3 block(BLOCK_SIZE, BLOCK_SIZE);
//   dim3 grid(S / BLOCK_SIZE, M / BLOCK_SIZE);
//   gpuMatMultWithTextureKernel<<<grid, block>>>(dev_result, M, N, S);

//   hipEventRecord(gpuFinish, 0);
//   hipEventSynchronize(gpuFinish);
//   hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
//   printf("\nThe runing time of GPU on Mat Multiply is %f seconds.\n", elapsedTime / 1000.0);

//   cudaStatus = hipGetLastError();
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));
//     goto Error;
//   }

//   cudaStatus = hipDeviceSynchronize();
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "hipDeviceSynchronize return Error code %d after Kernel launched!\n", cudaStatus);
//     goto Error;
//   }

//   cudaStatus = hipMemcpy(result, dev_result, M * S * sizeof(int), hipMemcpyDeviceToHost);
//   if (cudaStatus != hipSuccess)
//   {
//     fprintf(stderr, "hipMemcpy result failed!\n");
//     goto Error;
//   }

// Error:
//   hipUnbindTexture(texA);
//   hipUnbindTexture(texB);
//   hipFree(dev_a);
//   hipFree(dev_b);
//   hipFree(dev_result);

//   return cudaStatus;
// }